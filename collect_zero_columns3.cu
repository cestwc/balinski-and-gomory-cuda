#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// One thread per element version
__global__ void collectZeroIndicesElementWise(const int* B, int n,
                                              int* zero_indices,
                                              int* row_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n * n) return;

    int row = idx / n;
    int col = idx % n;

    if (B[idx] == 0) {
        int pos = atomicAdd(&row_count[row], 1);
        zero_indices[row * n + pos] = col;
    }
}


int main() {
    int n = 4096; // or large like 4096
    std::cout << "Matrix size: " << n << " x " << n << std::endl;

    std::vector<int> h_B(n * n);
    srand(0);
    for (int i = 0; i < n * n; ++i)
        h_B[i] = (rand() % 10 == 0) ? 0 : rand() % 100; // ~10% zeros

    int *d_B, *d_indices, *d_row_count;
    hipMalloc(&d_B, n * n * sizeof(int));
    hipMalloc(&d_indices, n * n * sizeof(int));
    hipMalloc(&d_row_count, n * sizeof(int));
    hipMemcpy(d_B, h_B.data(), n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_row_count, 0, n * sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int totalThreads = n * n;
    int blockSize = 256;
    int gridSize = (totalThreads + blockSize - 1) / blockSize;
    collectZeroIndicesElementWise<<<gridSize, blockSize>>>(d_B, n, d_indices, d_row_count);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Total GPU processing time: " << ms << " ms\n";

    if (n < 10) {
        std::vector<int> h_indices(n * n);
        std::vector<int> h_row_count(n);
        hipMemcpy(h_indices.data(), d_indices, n * n * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_row_count.data(), d_row_count, n * sizeof(int), hipMemcpyDeviceToHost);

        std::cout << "\nMatrix B:\n";
        for (int i = 0; i < n; ++i) {
            for (int j = 0; j < n; ++j)
                std::cout << h_B[i * n + j] << " ";
            std::cout << "\n";
        }

        std::cout << "\nZero column indices per row:\n";
        for (int i = 0; i < n; ++i) {
            std::cout << "Row " << i << " (count=" << h_row_count[i] << "): ";
            for (int k = 0; k < h_row_count[i]; ++k)
                std::cout << h_indices[i * n + k] << " ";
            std::cout << "\n";
        }
    }

    hipFree(d_B);
    hipFree(d_indices);
    hipFree(d_row_count);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
