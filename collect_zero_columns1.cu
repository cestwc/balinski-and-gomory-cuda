#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>

__global__ void countZerosPerRow(const int* B, int* zero_counts, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n) return;
    int count = 0;
    for (int j = 0; j < n; ++j)
        if (B[row * n + j] == 0) count++;
    zero_counts[row] = count;
}

__global__ void collectZeroIndices(const int* B, const int* row_offsets,
                                   int* zero_indices, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n) return;
    int start = row_offsets[row];
    int count = 0;
    for (int j = 0; j < n; ++j)
        if (B[row * n + j] == 0)
            zero_indices[start + count++] = j;
}

void prefixSum(const int* counts, int* offsets, int n) {
    offsets[0] = 0;
    for (int i = 1; i < n; ++i)
        offsets[i] = offsets[i-1] + counts[i-1];
}

int main() {
    int n = 4096; // change for testing
    std::cout << "Matrix size: " << n << " x " << n << std::endl;

    std::vector<int> h_B(n * n);
    std::srand((unsigned)time(0));
    for (int i = 0; i < n * n; ++i)
        h_B[i] = (std::rand() % 10 == 0) ? 0 : std::rand() % 100;

    int *d_B, *d_counts, *d_offsets, *d_indices;
    hipMalloc(&d_B, n*n*sizeof(int));
    hipMalloc(&d_counts, n*sizeof(int));
    hipMemcpy(d_B, h_B.data(), n*n*sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int blockSize = 256, gridSize = (n + blockSize - 1) / blockSize;
    countZerosPerRow<<<gridSize, blockSize>>>(d_B, d_counts, n);
    hipDeviceSynchronize();

    std::vector<int> h_counts(n), h_offsets(n);
    hipMemcpy(h_counts.data(), d_counts, n*sizeof(int), hipMemcpyDeviceToHost);
    prefixSum(h_counts.data(), h_offsets.data(), n);
    int totalZeros = h_offsets[n-1] + h_counts[n-1];

    hipMalloc(&d_offsets, n*sizeof(int));
    hipMalloc(&d_indices, totalZeros*sizeof(int));
    hipMemcpy(d_offsets, h_offsets.data(), n*sizeof(int), hipMemcpyHostToDevice);

    collectZeroIndices<<<gridSize, blockSize>>>(d_B, d_offsets, d_indices, n);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    std::cout << "Total GPU processing time: " << ms << " ms" << std::endl;

    // optional print for small n
    if (n < 10) {
        std::vector<int> h_indices(totalZeros);
        hipMemcpy(h_indices.data(), d_indices, totalZeros*sizeof(int), hipMemcpyDeviceToHost);

        std::cout << "Matrix B:\n";
        for (int i = 0; i < n; ++i) {
            for (int j = 0; j < n; ++j) std::cout << h_B[i*n + j] << " ";
            std::cout << "\n";
        }
        std::cout << "\nZero column indices per row:\n";
        for (int i = 0; i < n; ++i) {
            std::cout << "Row " << i << " (count=" << h_counts[i] << "): ";
            for (int k = 0; k < h_counts[i]; ++k)
                std::cout << h_indices[h_offsets[i] + k] << " ";
            std::cout << "\n";
        }
    }

    hipFree(d_B);
    hipFree(d_counts);
    hipFree(d_offsets);
    hipFree(d_indices);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
