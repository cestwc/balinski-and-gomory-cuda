#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>

__global__ void collectZeroIndicesSingleKernel(const int* B, int n,
                                               int* zero_indices,
                                               int* row_start,
                                               int* row_count,
                                               int* global_counter)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n) return;

    int start_pos = atomicAdd(global_counter, n);
    int count = 0;
    for (int j = 0; j < n; ++j)
        if (B[row * n + j] == 0)
            zero_indices[start_pos + count++] = j;

    row_start[row] = start_pos;
    row_count[row] = count;
}

int main() {
    int n = 4096;
    std::cout << "Matrix size: " << n << " x " << n << std::endl;

    std::vector<int> h_B(n * n);
    std::srand((unsigned)time(0));
    for (int i = 0; i < n * n; ++i)
        h_B[i] = (std::rand() % 10 == 0) ? 0 : std::rand() % 100;

    int *d_B, *d_indices, *d_row_start, *d_row_count, *d_counter;
    hipMalloc(&d_B, n*n*sizeof(int));
    hipMalloc(&d_indices, n*n*sizeof(int));
    hipMalloc(&d_row_start, n*sizeof(int));
    hipMalloc(&d_row_count, n*sizeof(int));
    hipMalloc(&d_counter, sizeof(int));
    hipMemcpy(d_B, h_B.data(), n*n*sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_counter, 0, sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int blockSize = 256, gridSize = (n + blockSize - 1) / blockSize;
    collectZeroIndicesSingleKernel<<<gridSize, blockSize>>>(
        d_B, n, d_indices, d_row_start, d_row_count, d_counter);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    std::cout << "Total GPU processing time: " << ms << " ms" << std::endl;

    // optional print for small n
    if (n < 10) {
        std::vector<int> h_indices(n*n);
        std::vector<int> h_row_start(n), h_row_count(n);
        hipMemcpy(h_indices.data(), d_indices, n*n*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_row_start.data(), d_row_start, n*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_row_count.data(), d_row_count, n*sizeof(int), hipMemcpyDeviceToHost);

        std::cout << "Matrix B:\n";
        for (int i = 0; i < n; ++i) {
            for (int j = 0; j < n; ++j) std::cout << h_B[i*n + j] << " ";
            std::cout << "\n";
        }
        std::cout << "\nZero column indices per row:\n";
        for (int i = 0; i < n; ++i) {
            std::cout << "Row " << i << " (count=" << h_row_count[i] << "): ";
            for (int k = 0; k < h_row_count[i]; ++k)
                std::cout << h_indices[h_row_start[i] + k] << " ";
            std::cout << "\n";
        }
    }

    hipFree(d_B);
    hipFree(d_indices);
    hipFree(d_row_start);
    hipFree(d_row_count);
    hipFree(d_counter);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
