#include <hip/hip_runtime.h>
// #include <iostream>
// #include <cstdlib>
// #include <ctime>

// CUDA kernel for matrix multiplication
__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < K) {
        float sum = 0.0f;
        for (int i = 0; i < N; ++i) {
            sum += A[row * N + i] * B[i * K + col];
        }
        C[row * K + col] = sum;
    }
}

#include <iostream>
#include <cstdlib>
#include <ctime>


// Forward declaration
// void solve(float* d_C, int* d_X, float* d_U, float* d_V, int n);


#include <iostream>
#include <float.h>

#define IDX2C(i,j,n) ((j)*(n)+(i))
// #define IDX2C(i,j,n) ((i)*(n)+(j))

// Device kernels
__global__ void compute_B(const float* C, const float* U, const float* V, float* B, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        B[IDX2C(i, j, n)] = C[IDX2C(i, j, n)] - U[i] - V[j];
    }
}

__global__ void find_argmin(const float* B, int* out_idx, float* out_val, int n) {
    __shared__ float min_val[256];
    __shared__ int min_idx[256];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    int total = n * n;

    float val = (index < total) ? B[index] : FLT_MAX;
    min_val[tid] = val;
    min_idx[tid] = index;
    __syncthreads();

    // Parallel reduction to find min
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && min_val[tid + s] < min_val[tid]) {
            min_val[tid] = min_val[tid + s];
            min_idx[tid] = min_idx[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out_val[blockIdx.x] = min_val[0];
        out_idx[blockIdx.x] = min_idx[0];
    }
}


__global__ void compute_col_to_row(int n, const int* X, int* col_to_row) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j >= n) return;

    for (int i = 0; i < n; ++i) {
        if (X[IDX2C(i, j, n)] == 1) {
            col_to_row[j] = i;
            return;
        }
    }
}

__global__ void solve_1bc_kernel(
    int n,
    const int* col_to_row,
    int k,
    const float* B,
    int* R,
    int* Q,
    bool* changed
) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // rows
    int j = blockIdx.x * blockDim.x + threadIdx.x; // columns

    if (i >= n || j >= n) return;

    // Step (b): still only run one thread per column
    if (i == 0 && Q[j] != n) {
        int row = col_to_row[j];
        if (R[row] == n) {
            R[row] = j;
            *changed = true;
        }
    }

    // Step (c): one thread per (i, j)
    if (i != k && R[i] != n && Q[j] == n) {
        float b_val = B[IDX2C(i, j, n)];
        if (b_val == 0.0f) {
            if (atomicMin(&Q[j], i) > i) {
                *changed = true;
            }
            // if (atomicCAS(&Q[j], n, i) == n) {
            //     *changed = true;
            // }
            // if (Q[j] == n) {
            //     Q[j] = i;
            //     *changed = true;
            // }
        }
    }
}


void solve_1bc(
    int n,
    int* d_col_to_row,
    int k,
    int l,
    float* d_B,
    int* d_R,
    int* d_Q
){

    dim3 threadsPerBlock(16, 16); // 16x16 = 256 threads per block
    dim3 numBlocks((n + 15) / 16, (n + 15) / 16); // ceil(n / 16) in each dimension

    bool h_changed;
    bool* d_changed;
    hipMalloc(&d_changed, sizeof(bool));

    do {
        h_changed = false;
        hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);
        solve_1bc_kernel<<<numBlocks, threadsPerBlock>>>(
            n, d_col_to_row, k, d_B, d_R, d_Q, d_changed
        );
        hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    } while (h_changed);

    hipFree(d_changed);
}


__global__ void set_array_value(int* arr, int value, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) arr[idx] = value;
}

__global__ void update_duals(int* R, int* Q, float* U, float* V, float epsilon, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        if (R[i] != n) U[i] += epsilon;
        if (Q[i] != n) V[i] -= epsilon;
    }
}

bool solve_from_kl(
    int n,
    float* d_C, int* d_X, int k, int l,
    float* d_U, float* d_V, float* d_B
) {
    // Allocate and initialize R and Q
    int* d_R; int* d_Q;
    hipMalloc(&d_R, n * sizeof(int));
    hipMalloc(&d_Q, n * sizeof(int));

    set_array_value<<<(n + 255)/256, 256>>>(d_R, n, n);
    set_array_value<<<(n + 255)/256, 256>>>(d_Q, n, n);

    // Q[l] = k
    hipMemcpy(&d_Q[l], &k, sizeof(int), hipMemcpyHostToDevice);

    // Step 1: Solve 1BC
    int* d_col_to_row;
    hipMalloc(&d_col_to_row, n * sizeof(int));
    compute_col_to_row<<<(n + 255) / 256, 256>>>(n, d_X, d_col_to_row);
    hipDeviceSynchronize();

    for (int s = 0; s < n; ++s) {
        solve_1bc(n, d_col_to_row, k, l, d_B, d_R, d_Q);
    }
    hipFree(d_col_to_row);

    // Step 2: Check if R[k] != n and R[k] != l
    int h_Rk;
    hipMemcpy(&h_Rk, &d_R[k], sizeof(int), hipMemcpyDeviceToHost);

    if (h_Rk != n && h_Rk != l) {
        int k_ = k;
        int l_ = l;

        int h_R, h_Q;

        while (true) {
            // X[k_, l_] = 1
            int one = 1;
            int idx_on = IDX2C(k_, l_, n);
            hipMemcpy(&d_X[idx_on], &one, sizeof(int), hipMemcpyHostToDevice);

            // l_ = R[k_]
            hipMemcpy(&h_R, &d_R[k_], sizeof(int), hipMemcpyDeviceToHost);
            l_ = h_R;

            // X[k_, l_] = 0
            int zero = 0;
            int idx_off = IDX2C(k_, l_, n);
            hipMemcpy(&d_X[idx_off], &zero, sizeof(int), hipMemcpyHostToDevice);

            // k_ = Q[l_]
            hipMemcpy(&h_Q, &d_Q[l_], sizeof(int), hipMemcpyDeviceToHost);
            k_ = h_Q;

            if (k_ == k && l_ == l)
                break;
        }
        

        float b_kl;
        hipMemcpy(&b_kl, &d_B[IDX2C(k, l, n)], sizeof(float), hipMemcpyDeviceToHost);
        float epsilon = -b_kl;

        float v_l;
        hipMemcpy(&v_l, &d_V[l], sizeof(float), hipMemcpyDeviceToHost);
        v_l -= epsilon;
        hipMemcpy(&d_V[l], &v_l, sizeof(float), hipMemcpyHostToDevice);

        // Recompute B = C - U.unsqueeze(1) - V
        dim3 threads(16, 16);
        dim3 blocks((n + 15) / 16, (n + 15) / 16);
        compute_B<<<blocks, threads>>>(d_C, d_U, d_V, d_B, n);

        float* h_B = new float[n * n];
        hipMemcpy(h_B, d_B, sizeof(float) * n * n, hipMemcpyDeviceToHost);

        int min_idx = 0;
        float min_val = h_B[0];
        for (int idx = 1; idx < n * n; ++idx) {
            if (h_B[idx] < min_val) {
                min_val = h_B[idx];
                min_idx = idx;
            }
        }
        delete[] h_B;

        // Update k, l in-place
        k = min_idx % n;
        l = min_idx / n;

        // cudaFree(d_R);
        // cudaFree(d_Q);
        // return false;
        hipFree(d_R);
        hipFree(d_Q);
        return solve_from_kl(n, d_C, d_X, k, l, d_U, d_V, d_B);  // recursion

    }

    // Branch B: find epsilon satisfying mask
    float epsilon = -1;
    bool found = false;
    // int i_found = -1, j_found = -1;

    for (int i = 0; i < n; ++i) {
        int r_val;
        hipMemcpy(&r_val, &d_R[i], sizeof(int), hipMemcpyDeviceToHost);
        if (r_val == n) continue;

        for (int j = 0; j < n; ++j) {
            int q_val;
            hipMemcpy(&q_val, &d_Q[j], sizeof(int), hipMemcpyDeviceToHost);
            if (q_val != n) continue;

            float b_ij;
            hipMemcpy(&b_ij, &d_B[IDX2C(i, j, n)], sizeof(float), hipMemcpyDeviceToHost);
            if (b_ij >= 0 && (!found || b_ij < epsilon)) {
                epsilon = b_ij;
                // i_found = i;
                // j_found = j;
                found = true;
            }
        }
    }

    if (!found) {
        float b_kl;
        hipMemcpy(&b_kl, &d_B[IDX2C(k, l, n)], sizeof(float), hipMemcpyDeviceToHost);
        epsilon = -b_kl;
    }

    // Update duals
    update_duals<<<(n + 255) / 256, 256>>>(d_R, d_Q, d_U, d_V, epsilon, n);

    // Recompute B = C - U.unsqueeze(1) - V
    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (n + 15) / 16);
    compute_B<<<blocks, threads>>>(d_C, d_U, d_V, d_B, n);
    hipDeviceSynchronize();

    // Check B[k,l]
    float b_kl_check;
    hipMemcpy(&b_kl_check, &d_B[IDX2C(k, l, n)], sizeof(float), hipMemcpyDeviceToHost);

    if (b_kl_check < 0) {
        // cudaFree(d_R);
        // cudaFree(d_Q);
        // return true;
        hipFree(d_R);
        hipFree(d_Q);
        return solve_from_kl(n, d_C, d_X, k, l, d_U, d_V, d_B);  // recursion

    }

    // Check if any B[i,j] < 0
    bool any_negative = false;
    for (int i = 0; i < n && !any_negative; ++i) {
        for (int j = 0; j < n; ++j) {
            float b_ij;
            hipMemcpy(&b_ij, &d_B[IDX2C(i, j, n)], sizeof(float), hipMemcpyDeviceToHost);
            if (b_ij < 0) {
                any_negative = true;
                // break;
                hipFree(d_R);
                hipFree(d_Q);
                return solve_from_kl(n, d_C, d_X, i, j, d_U, d_V, d_B);  // recursion
            }
        }
    }

    hipFree(d_R);
    hipFree(d_Q);
    return any_negative;
}



void solve(float* d_C, int* d_X, float* d_U, float* d_V, int n) {
    size_t sizeMat = n * n * sizeof(float);

    // Allocate B
    float* d_B;
    hipMalloc(&d_B, sizeMat);

    // Allocate buffers for argmin
    int* d_idx; float* d_val;
    hipMalloc(&d_idx, sizeof(int));
    hipMalloc(&d_val, sizeof(float));

    // Compute initial B
    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (n + 15) / 16);
    compute_B<<<blocks, threads>>>(d_C, d_U, d_V, d_B, n);

    int steps = 0;
    while (true) {
        // Find argmin(B)
        int totalThreads = n * n;
        int blockSize = 256;
        int gridSize = (totalThreads + blockSize - 1) / blockSize;
        find_argmin<<<gridSize, blockSize>>>(d_B, d_idx, d_val, n);

        int h_idx;
        float h_val;
        hipMemcpy(&h_idx, d_idx, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_val, d_val, sizeof(float), hipMemcpyDeviceToHost);

        int k = h_idx / n;
        int l = h_idx % n;

        // std::cout << "Step " << steps << ": argmin at B[" << k << "][" << l << "] = " << h_val << "\n";

        // Call solve_from_kl, which returns false if we should stop
        bool should_continue = solve_from_kl(n, d_C, d_X, k, l, d_U, d_V, d_B);
        steps++;

        if (!should_continue) {
            std::cout << "Solver has converged after " << steps << " steps.\n";
            break;
        }
    }

    // Cleanup
    hipFree(d_B);
    hipFree(d_idx);
    hipFree(d_val);
}


__global__ void check_feasible_condition(const float* C, const int* X, const float* U, const float* V, int* out, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        if (X[IDX2C(i, j, n)] == 1) {
            float diff = C[IDX2C(i, j, n)] - U[i] - V[j];
            if (fabsf(diff) > 1e-4f) {
                atomicExch(out, 1); // feasible violated
            }
        }
    }
}

__global__ void check_slack_condition(const float* C, const float* U, const float* V, int* out, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        float diff = C[IDX2C(i, j, n)] - U[i] - V[j];
        if (diff < -1e-4f) {
            atomicExch(out, 1); // slack violated
        }
    }
}

void verify_solution(float* d_C, int* d_X, float* d_U, float* d_V, int n) {
    int h_feasible = 0, h_slack = 0;
    int *d_feasible, *d_slack;
    hipMalloc(&d_feasible, sizeof(int));
    hipMalloc(&d_slack, sizeof(int));
    hipMemset(d_feasible, 0, sizeof(int));
    hipMemset(d_slack, 0, sizeof(int));

    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (n + 15) / 16);

    check_feasible_condition<<<blocks, threads>>>(d_C, d_X, d_U, d_V, d_feasible, n);
    check_slack_condition<<<blocks, threads>>>(d_C, d_U, d_V, d_slack, n);

    hipMemcpy(&h_feasible, d_feasible, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_slack, d_slack, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_feasible);
    hipFree(d_slack);

    bool feasible_ok = (h_feasible == 0);
    bool slack_ok = (h_slack == 0);

    std::cout << "\n=== Post-Solution Verification ===\n";
    std::cout << "feasible condition: " << (feasible_ok ? "PASS" : "FAIL") << "\n";
    std::cout << "slack condition: " << (slack_ok ? "PASS" : "FAIL") << "\n";
    std::cout << "Overall check: " << ((feasible_ok && slack_ok) ? "✓ Passed" : "✗ Failed") << "\n\n";
}