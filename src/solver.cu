#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <float.h>
#include <hipcub/hipcub.hpp>
#include <string>
#include <cstdio>
#include <type_traits>
#include <hip/hip_math_constants.h>

template <typename T>
void printDeviceVar(const char* name, const T& symbol) {
    T host_val;
    hipMemcpyFromSymbol(&host_val, HIP_SYMBOL(symbol), sizeof(T), 0, hipMemcpyDeviceToHost);
    std::cout << name << " = " << host_val << std::endl;
}

template<typename T>
void printDeviceMatrix(const char* name, const T* d_M, int n) {
    T* h_M = new T[n*n];
    hipMemcpy(h_M, d_M, n*n*sizeof(T), hipMemcpyDeviceToHost);
    printf("%s (%dx%d):\n", name, n, n);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            int idx = j*n + i;
            if constexpr (std::is_floating_point<T>::value)
                printf("%6.2f ", static_cast<double>(h_M[idx]));
            else
                printf("%4d ", static_cast<int>(h_M[idx]));
        }
        printf("\n");
    }
    printf("\n");
    delete[] h_M;
}

template<typename T>
void printDeviceVector(const char* name, const T* d_V, int n) {
    T* h_V = new T[n];
    hipMemcpy(h_V, d_V, n*sizeof(T), hipMemcpyDeviceToHost);
    printf("%s (len=%d): ", name, n);
    for (int i = 0; i < n; i++) {
        if constexpr (std::is_floating_point<T>::value)
            printf("%6.2f ", static_cast<double>(h_V[i]));
        else
            printf("%d ", static_cast<int>(h_V[i]));
    }
    printf("\n\n");
    delete[] h_V;
}

template<typename T>
void printDeviceScalar(const char* name, const T* d_val) {
    T h_val;
    hipMemcpy(&h_val, d_val, sizeof(T), hipMemcpyDeviceToHost);
    if constexpr (std::is_floating_point<T>::value)
        printf("%s = %f\n\n", name, static_cast<double>(h_val));
    else
        printf("%s = %d\n\n", name, static_cast<int>(h_val));
}

#define IDX2C(i,j,n) ((j)*(n)+(i))

__global__ void compute_B(const float* C, const float* U, const float* V, float* B, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && j < n) B[IDX2C(i, j, n)] = C[IDX2C(i, j, n)] - U[i] - V[j];
}

__global__ void find_argmin(const float* B, int* out_idx, float* out_val, int n) {
    __shared__ float min_val[256];
    __shared__ int min_idx[256];
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    int total = n * n;
    float val = (index < total) ? B[index] : FLT_MAX;
    min_val[tid] = val;
    min_idx[tid] = index;
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && min_val[tid + s] < min_val[tid]) {
            min_val[tid] = min_val[tid + s];
            min_idx[tid] = min_idx[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        out_val[blockIdx.x] = min_val[0];
        out_idx[blockIdx.x] = min_idx[0];
    }
}

__device__  float d_min;
__device__  int d_changed;
__device__  float d_epsilon;
__device__  int d_found;
__device__  int d_flag;
__device__  int d_b_kl_neg;

__global__ void solve_1bc_kernel(int n, const int* X, int* k, int* l, const float* B, int* R, int* Q) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n || j >= n) return;
    if (Q[j] != n && X[IDX2C(i, j, n)] == 1) {
        if (atomicCAS(&R[i], n, j) == n) d_changed = 1;
    }
    if (i != *k && R[i] != n && Q[j] == n) {
        float b_val = B[IDX2C(i, j, n)];
        if (b_val == 0.0f) {
            if (atomicMin(&Q[j], i) > i) d_changed = 1;
        }
    }
}

__global__ void set_array_value(int* arr, int value, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) arr[idx] = value;
}

__global__ void update_duals(int* R, int* Q, float* U, float* V, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        if (R[i] != n) U[i] += d_epsilon;
        if (Q[i] != n) V[i] -= d_epsilon;
    }
}

__device__ inline void atomicMinFloatNonNeg(float* addr, float val) {
    atomicMin(reinterpret_cast<unsigned int*>(addr), __float_as_uint(val));
}

__global__ void init_minval() {
    d_min = HIP_INF_F;
}

__device__ float atomicMinFloat(float* addr, float value) {
    int* addr_as_int = (int*)addr;
    int old = *addr_as_int, assumed;
    do {
        assumed = old;
        float f_old = __int_as_float(assumed);
        if (f_old <= value) break;
        old = atomicCAS(addr_as_int, assumed, __float_as_int(value));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void find_most_negative(const float* __restrict__ d_B, int n, int* d_out_i, int* d_out_j) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    int threads_per_block = blockDim.x * blockDim.y;
    extern __shared__ float s_vals[];
    __shared__ int s_rows[256];
    __shared__ int s_cols[256];
    float val = INFINITY;
    int myRow = -1, myCol = -1;
    if (row < n && col < n) {
        float tmp = d_B[IDX2C(row, col, n)];
        if (tmp < 0.0f) {
            val = tmp;
            myRow = row;
            myCol = col;
        }
    }
    s_vals[tid] = val;
    s_rows[tid] = myRow;
    s_cols[tid] = myCol;
    __syncthreads();
    for (int stride = threads_per_block >> 1; stride > 0; stride >>= 1) {
        if (tid < stride) {
            if (s_vals[tid + stride] < s_vals[tid]) {
                s_vals[tid] = s_vals[tid + stride];
                s_rows[tid] = s_rows[tid + stride];
                s_cols[tid] = s_cols[tid + stride];
            }
        }
        __syncthreads();
    }
    if (tid == 0 && s_vals[0] < INFINITY) {
        d_found = 1;
        float oldMin = atomicMinFloat(&d_min, s_vals[0]);
        if (s_vals[0] < oldMin) {
            *d_out_i = s_rows[0];
            *d_out_j = s_cols[0];
        }
    }
}

__global__ void find_min_valid_atomic2d(const float* __restrict__ d_B, const int* __restrict__ d_R, const int* __restrict__ d_Q, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    int threads_per_block = blockDim.x * blockDim.y;
    extern __shared__ float sdata[];
    float val = HIP_INF_F;
    if (row < n && col < n) {
        if (d_R[row] != n && d_Q[col] == n) {
            float tmp = d_B[IDX2C(row, col, n)];
            if (tmp >= 0.0f) val = tmp;
        }
    }
    sdata[tid] = val;
    __syncthreads();
    for (int stride = threads_per_block >> 1; stride > 0; stride >>= 1) {
        if (tid < stride) sdata[tid] = fminf(sdata[tid], sdata[tid + stride]);
        __syncthreads();
    }
    if (tid == 0) {
        float block_min = sdata[0];
        if (block_min < HIP_INF_F) atomicMinFloatNonNeg(&d_min, block_min);
    }
}

__global__ void process_cycle(float* B, float* V, int* d_X, const int* d_R, const int* d_Q, int n, int* k, int* l) {
    int k_ = *k;
    int l_ = *l;
    while (true) {
        d_X[IDX2C(k_, l_, n)] = 1;
        l_ = d_R[k_];
        d_X[IDX2C(k_, l_, n)] = 0;
        k_ = d_Q[l_];
        if (k_ == *k && l_ == *l) break;
    }
    V[*l] += B[IDX2C(*k, *l, n)];
}

__global__ void finalize_epsilon(const float* d_B, int n, int* k, int* l) {
    if (isinf(d_min)) d_epsilon = -d_B[IDX2C(*k, *l, n)];
    else d_epsilon = d_min;
}

__global__ void update_Q(int* d_Q, const int* k, const int* l) {
    d_Q[*l] = *k;
}

__global__ void reset_d_changed() {
    d_changed = 0;
}

__global__ void check_Rk(const int* d_R, const int* k, const int* l, int n) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int Rk = d_R[*k];
        d_flag = (Rk != n && Rk != *l);
    }
}

__global__ void check_bkl(const float* d_B, const int* k, const int* l, int n) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int row = *k;
        int col = *l;
        float b_kl = d_B[IDX2C(row, col, n)];
        d_b_kl_neg = (b_kl < 0.0f);
    }
}

__global__ void reset_d_found() {
    if (threadIdx.x == 0 && blockIdx.x == 0) d_found = 0;
}

bool solve_from_kl(float* d_C, int* d_X, float* d_U, float* d_V, int n, float* d_B, int* d_R, int* d_Q, int* k, int* l) {
    update_Q<<<1,1>>>(d_Q, k, l);
    dim3 threads(16, 16);
    dim3 blocks((n + threads.x - 1) / threads.x, (n + threads.y - 1) / threads.y);
    
    int h_changed;
    do {
        reset_d_changed<<<1,1>>>();
        
        solve_1bc_kernel<<<blocks, threads>>>(n, d_X, k, l, d_B, d_R, d_Q);
        
        hipMemcpyFromSymbol(&h_changed, HIP_SYMBOL(d_changed), sizeof(int), 0, hipMemcpyDeviceToHost);

    } while (h_changed == 1);
    check_Rk<<<1,1>>>(d_R, k, l, n);
    

    int h_flag;
    hipMemcpyFromSymbol(&h_flag, HIP_SYMBOL(d_flag), sizeof(int), 0, hipMemcpyDeviceToHost);

    if (h_flag == 1) {
        process_cycle<<<1,1>>>(d_B, d_V, d_X, d_R, d_Q, n, k, l);
        
        find_most_negative<<<blocks, threads>>>(d_B, n, k, l);
        set_array_value<<<(n + 255)/256, 256>>>(d_R, n, n);
        set_array_value<<<(n + 255)/256, 256>>>(d_Q, n, n);
        compute_B<<<blocks, threads>>>(d_C, d_U, d_V, d_B, n);
        return true;
    }
    init_minval<<<1, 1>>>();
    
    find_min_valid_atomic2d<<<blocks, threads>>>(d_B, d_R, d_Q, n);
    finalize_epsilon<<<1, 1>>>(d_B, n, k, l);
    update_duals<<<(n + 255) / 256, 256>>>(d_R, d_Q, d_U, d_V, n);
    compute_B<<<blocks, threads>>>(d_C, d_U, d_V, d_B, n);
    
    check_bkl<<<1,1>>>(d_B, k, l, n);
    

    int h_b_kl_neg;
    hipMemcpyFromSymbol(&h_b_kl_neg, HIP_SYMBOL(d_b_kl_neg), sizeof(int), 0, hipMemcpyDeviceToHost);

    if (h_b_kl_neg == 1) return true;
    reset_d_found<<<1,1>>>();
    
    find_most_negative<<<blocks, threads>>>(d_B, n, k, l);

    int h_found;
    hipMemcpyFromSymbol(&h_found, HIP_SYMBOL(d_found), sizeof(int), 0, hipMemcpyDeviceToHost);

    
    if (h_found) {
        set_array_value<<<(n + 255)/256, 256>>>(d_R, n, n);
        set_array_value<<<(n + 255)/256, 256>>>(d_Q, n, n);
        return true;
    } else {
        return false;
    }
}

void solve(float* d_C, int* d_X, float* d_U, float* d_V, int n) {
    float* d_B;
    hipMalloc(&d_B, n * n * sizeof(float));
    int *k, *l;
    hipMalloc(&k, sizeof(int));
    hipMalloc(&l, sizeof(int));
    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (n + 15) / 16);
    compute_B<<<blocks, threads>>>(d_C, d_U, d_V, d_B, n);
    int* d_R; int* d_Q;
    hipMalloc(&d_R, n * sizeof(int));
    hipMalloc(&d_Q, n * sizeof(int));
    set_array_value<<<(n + 255)/256, 256>>>(d_R, n, n);
    set_array_value<<<(n + 255)/256, 256>>>(d_Q, n, n);
    find_most_negative<<<blocks, threads>>>(d_B, n, k, l);
    int steps = 0;
    while (true) {
        // std::cout << "Step " << steps << " \n";
        bool should_continue = solve_from_kl(d_C, d_X, d_U, d_V, n, d_B, d_R, d_Q, k, l);
        steps++;
        if (!should_continue) {
            // std::cout << "Solver has converged after " << steps << " steps.\n";
            break;
        }
    }
    hipFree(d_B);
    hipFree(d_R);
    hipFree(d_Q);
    hipFree(k);
    hipFree(l);
}



__global__ void check_feasible_condition(const float* C, const int* X, const float* U, const float* V, int* out, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        if (X[IDX2C(i, j, n)] == 1) {
            float diff = C[IDX2C(i, j, n)] - U[i] - V[j];
            if (fabsf(diff) > 1e-4f) {
                atomicExch(out, 1); // feasible violated
            }
        }
    }
}

__global__ void check_slack_condition(const float* C, const float* U, const float* V, int* out, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        float diff = C[IDX2C(i, j, n)] - U[i] - V[j];
        if (diff < -1e-4f) {
            atomicExch(out, 1); // slack violated
        }
    }
}

void verify_solution(float* d_C, int* d_X, float* d_U, float* d_V, int n) {
    int h_feasible = 0, h_slack = 0;
    int *d_feasible, *d_slack;
    hipMalloc(&d_feasible, sizeof(int));
    hipMalloc(&d_slack, sizeof(int));
    hipMemset(d_feasible, 0, sizeof(int));
    hipMemset(d_slack, 0, sizeof(int));

    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (n + 15) / 16);

    check_feasible_condition<<<blocks, threads>>>(d_C, d_X, d_U, d_V, d_feasible, n);
    check_slack_condition<<<blocks, threads>>>(d_C, d_U, d_V, d_slack, n);

    hipMemcpy(&h_feasible, d_feasible, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_slack, d_slack, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_feasible);
    hipFree(d_slack);

    bool feasible_ok = (h_feasible == 0);
    bool slack_ok = (h_slack == 0);

    std::cout << "\n=== Post-Solution Verification ===\n";
    std::cout << "feasible condition: " << (feasible_ok ? "PASS" : "FAIL") << "\n";
    std::cout << "slack condition: " << (slack_ok ? "PASS" : "FAIL") << "\n";
    std::cout << "Overall check: " << ((feasible_ok && slack_ok) ? "✓ Passed" : "✗ Failed") << "\n\n";
}